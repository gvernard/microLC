#include <cstring>
#include <cstdio>

#include "data_structs.hpp"



Chi2Vars::Chi2Vars(int Njp): Njp(Njp) {
  this->indA  = (int*) malloc(this->Njp*sizeof(int));
  this->indB  = (int*) malloc(this->Njp*sizeof(int));
  this->facA  = (double*) malloc(this->Njp*sizeof(double));
  this->facB  = (double*) malloc(this->Njp*sizeof(double));
  this->new_d = (double*) malloc(this->Njp*sizeof(double));
  this->new_s = (double*) malloc(this->Njp*sizeof(double));
}

Chi2Vars::Chi2Vars(const Chi2Vars& other): Chi2Vars(other.Njp){
  std::memcpy(this->indA,other.indA,this->Njp*sizeof(int));
  std::memcpy(this->indB,other.indB,this->Njp*sizeof(int));
  std::memcpy(this->facA,other.facA,this->Njp*sizeof(double));
  std::memcpy(this->facB,other.facB,this->Njp*sizeof(double));
  std::memcpy(this->new_d,other.new_d,this->Njp*sizeof(double));
  std::memcpy(this->new_s,other.new_s,this->Njp*sizeof(double));
}

Chi2Vars::~Chi2Vars(){
  free(indA);
  free(indB);
  free(facA);
  free(facB);
  free(new_d);
  free(new_s);
}



Chi2::Chi2(int Nloc): Nloc(Nloc){
  this->values = (double*) malloc(this->Nloc*this->Nloc*sizeof(double));
  hipMalloc(&this->d_values,this->Nloc*this->Nloc*sizeof(double));
  printf("Allocated memory: for chi2 of all pairs of light curves, with size Nloc*Nloc <double>: %ld (bytes)\n",Nloc*Nloc*sizeof(double));
}

Chi2::Chi2(const Chi2& other): Chi2(other.Nloc){
  // The copy constructor is not expected to be used
  std::memcpy(this->values,other.values,this->Nloc*this->Nloc*sizeof(double));
  hipMemcpy(this->d_values,other.d_values,this->Nloc*this->Nloc*sizeof(double),hipMemcpyDeviceToDevice);
}

Chi2::~Chi2(){
  free(values);
  hipFree(d_values);
}

void Chi2::transfer_to_CPU(){
  hipMemcpy(this->values,this->d_values,this->Nloc*this->Nloc*sizeof(double),hipMemcpyDeviceToHost);
}



SimLC::SimLC(int Nloc,int Nprof): Nloc(Nloc),Nprof(Nprof){
  this->LC  = (double*) malloc(this->Nloc*this->Nprof*sizeof(double));
  this->DLC = (double*) malloc(this->Nloc*(this->Nprof-1)*sizeof(double));
  hipMalloc(&this->d_LC,this->Nloc*this->Nprof*sizeof(double));
  hipMalloc(&this->d_DLC,this->Nloc*(this->Nprof-1)*sizeof(double));
  printf("%d %d\n",this->Nprof,this->Nloc);
  printf("Allocated memory: Light curves, (2Nprof-1)*Nloc <double>: %ld (Mb)\n",(2*this->Nprof-1)*this->Nloc*sizeof(double));
};

SimLC::SimLC(const SimLC& other): SimLC(other.Nloc,other.Nprof){
  // The copy constructor is not expected to be used
  std::memcpy(this->LC,other.LC,this->Nloc*this->Nprof*sizeof(double));
  std::memcpy(this->DLC,other.DLC,this->Nloc*(this->Nprof-1)*sizeof(double));
  hipMemcpy(this->d_LC,other.d_LC,this->Nloc*this->Nprof*sizeof(double),hipMemcpyDeviceToDevice);
  hipMemcpy(this->d_DLC,other.d_DLC,this->Nloc*(this->Nprof-1)*sizeof(double),hipMemcpyDeviceToDevice);
};

SimLC::~SimLC(){
  free(LC);
  free(DLC);
  hipFree(d_LC);
  hipFree(d_DLC);
}

void SimLC::transfer_to_CPU(){
  hipMemcpy(this->LC,this->d_LC,this->Nloc*this->Nprof*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(this->DLC,this->d_DLC,this->Nloc*(this->Nprof-1)*sizeof(double),hipMemcpyDeviceToHost);
}