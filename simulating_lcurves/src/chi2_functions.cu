#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <iostream>

#include "chi2_functions.hpp"
#include "util.hpp"

__constant__ double d_d[50];
__constant__ double d_s[50];
__constant__ double d_facA[50];
__constant__ double d_facB[50];
__constant__ int d_indA[50];
__constant__ int d_indB[50];

__global__ void kernelChi2(int loop_ind,int N,int Nprof,double* LCA,double* LCB,double* DLCA,double* DLCB,int Nloc,double* chi2_all,double* z_all);


__global__ void kernelChi2(int loop_ind,int N,int Nprof,double* LCA,double* LCB,double* DLCA,double* DLCB,int Nloc,double* chi2_all,double* z_all){
  int Nt  = blockDim.x; // equal to 1024
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int a = bid*Nt + tid;
  double chi2,mB,tmp;
  int b;

  // Get all the values of mA just once for each thread
  double mA[50];
  for(int i=0;i<N;i++){
    // Interpolate magnification A
    if( d_indA[i] == -1 ){
      mA[i] = 1;
    } else {
      //mA[i] = LCA[a*Nprof+d_indA[i]] + d_facA[i]*( LCA[a*Nprof+d_indA[i]+1] - LCA[a*Nprof+d_indA[i]] );
      mA[i] = LCA[a*Nprof+d_indA[i]] + d_facA[i]*DLCA[a*Nprof+d_indA[i]];
    }
  }
  __syncthreads();


  for(int j=0;j<Nt;j++){
    b = ((bid+loop_ind)%gridDim.x)*Nt + (tid+j)%Nt;

    chi2 = 0.0;
    for(int i=0;i<N;i++){
      // Interpolate magnification B
      if( d_indB[i] == -1 ){
	mB = 1;
      } else {
	//mB = LCB[b*Nprof+d_indB[i]] + d_facB[i]*( LCB[b*Nprof+d_indB[i]+1] - LCB[b*Nprof+d_indB[i]] );
	mB = LCB[b*Nprof+d_indB[i]] + d_facB[i]*DLCB[b*Nprof+d_indB[i]];
      }
      
      // Calculate chi2 term
      tmp = (d_d[i] - (mA[i]/mB))/d_s[i];
      chi2 += tmp*tmp;
    }
    chi2_all[a*Nloc+b] = chi2;

    z_all[a*Nloc+b] = LCA[a*Nprof]/LCB[b*Nprof];
    
    __syncthreads();
  }

}


double calculate_chi2_GPU(int N,int* indA,int* indB,double* facA,double* facB,double* d,double* s,int Nloc,int Nprof,double* LCA,double* LCB,double* DLCA,double* DLCB){  

  // ############################################## Memory Allocation on the GPU ##############################################
  printf("Used GPU memory:  %d (Mb)\n",(int) get_used_gpu_mem());

  
  // Transfer the fixed arrays to the GPU
  hipMemcpyToSymbol(HIP_SYMBOL(d_d),d,N*sizeof(double),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_s),s,N*sizeof(double),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_facA),facA,N*sizeof(double),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_facB),facB,N*sizeof(double),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_indA),indA,N*sizeof(int),0,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(d_indB),indB,N*sizeof(int),0,hipMemcpyHostToDevice);
  printf("Allocated CONSTANT memory: all arrays of size %d - data <double>, uncertainty <double>, factors <double> (x2), and indices <int> (x2): %ld (bytes)\n",N,(4*sizeof(double)+2*sizeof(int))*N);

  // Transfer LC for both images to the GPU
  double* d_LCA;
  hipMalloc(&d_LCA,Nloc*Nprof*sizeof(double));
  hipMemcpy(d_LCA,LCA,Nloc*Nprof*sizeof(double),hipMemcpyHostToDevice);
  double* d_LCB;
  hipMalloc(&d_LCB,Nloc*Nprof*sizeof(double));
  hipMemcpy(d_LCB,LCB,Nloc*Nprof*sizeof(double),hipMemcpyHostToDevice);
  double* d_DLCA;
  hipMalloc(&d_DLCA,Nloc*(Nprof-1)*sizeof(double));
  hipMemcpy(d_DLCA,DLCA,Nloc*(Nprof-1)*sizeof(double),hipMemcpyHostToDevice);
  double* d_DLCB;
  hipMalloc(&d_DLCB,Nloc*(Nprof-1)*sizeof(double));
  hipMemcpy(d_DLCB,DLCB,Nloc*(Nprof-1)*sizeof(double),hipMemcpyHostToDevice);
  printf("Allocated memory: LCs (x2) and DLCs (x2), with size Nloc*Nprof and Nloc*(Nprof-1) <double>: %ld (bytes)\n",(4*Nprof-2)*Nloc*sizeof(double));

  // Allocate memory for chi2 and z values
  double* d_chi2;
  hipMalloc(&d_chi2,Nloc*Nloc*sizeof(double));
  double* d_z;
  hipMalloc(&d_z,Nloc*Nloc*sizeof(double));
  printf("Allocated memory: for chi2 and z of all pairs of light curves, with size 2*Nloc*Nloc <double>: %ld (bytes)\n",2*Nloc*Nloc*sizeof(double));

  
  printf("Used GPU memory:  %d (Mb)\n",(int) get_used_gpu_mem());
  // ##########################################################################################################################



  
  // Call chi2 kernel
  int Nblocks = (int) ceil(Nloc/1024);
  for(int k=0;k<Nblocks;k++){
    kernelChi2<<<Nblocks,1024>>>(k,N,Nprof,d_LCA,d_LCB,d_DLCA,d_DLCB,Nloc,d_chi2,d_z);
  }
  hipFree(d_LCA);
  hipFree(d_LCB);
  hipFree(d_DLCA);
  hipFree(d_DLCB);



  // Fetch chi2 terms
  // Print a few of them
  double* test_chi2 = (double*) malloc(10*sizeof(double));
  int index = 23*Nloc+24;
  hipMemcpy(test_chi2,d_chi2+index,10*sizeof(double),hipMemcpyDeviceToHost);
  for(int i=0;i<10;i++){
    std::cout << test_chi2[i] << std::endl;
  }
  free(test_chi2);
  std::cout << std::endl;

  
  // Bin exp(-1/2 chi2) by z

  // Loop over z bins and multiply the binned chi2 with the z prior value 

  // Loop over z bins and perform the integration
  double integral = 1.0;
  

  hipFree(d_chi2);
  hipFree(d_z);  
  return log(integral);
}



double calculate_chi2_CPU(int N,int* indA,int* indB,double* facA,double* facB,double* d,double* s,int Nloc,int Nprof,double* LCA,double* LCB,double* DLCA,double* DLCB){
  double* chi2_all = (double*) malloc(Nloc*Nloc*sizeof(double));
  double mA,mB,tmp,chi2;
  

  for(int a=0;a<Nloc;a++){
    for(int b=0;b<Nloc;b++){

      chi2 = 0.0;
      for(int i=0;i<N;i++){
	if( indA[i] == -1 ){
	  mA = 1;
	} else {
	  mA = LCA[a*Nprof+indA[i]] + facA[i]*DLCA[a*Nprof+indA[i]];
	}
	
	if( indB[i] == -1 ){
	  mB = 1;
	} else {
	  mB = LCB[b*Nprof+indB[i]] + facB[i]*DLCB[b*Nprof+indB[i]];
	}
	
	tmp = (d[i] - (mA/mB))/s[i];
	chi2 += tmp*tmp;
      }
      chi2_all[a*Nloc+b] = chi2; 

    }
  }


  int index = 23*Nloc+24;
  for(int i=0;i<10;i++){
    std::cout << chi2_all[index+i] << std::endl;
  }

  free(chi2_all);
  double integral = 1.0;
  return log(integral);
}
