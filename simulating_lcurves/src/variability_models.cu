#include "hip/hip_runtime.h"
#include <cmath>
#include <hipfft/hipfft.h>
#include <iostream>

#include "variability_models.hpp"
#include "magnification_map.hpp"
#include "profile.hpp"
#include "util.hpp"


static int myfft2d_r2c(hipfftHandle* plan, hipfftDoubleReal* data, hipfftDoubleComplex* Fdata);
static int myfft2d_c2r(hipfftHandle* plan, hipfftDoubleComplex* Fdata, hipfftDoubleReal* data);
__global__ void kernelMultiplyFFTs(hipfftDoubleComplex* Fmap,hipfftDoubleComplex* Fkernel,double norm);
__global__ void sampleConvmap(hipfftDoubleReal* convmap,double* LC,double* DLC,int k,int Nprof,int* loc_x,int* loc_y,int Nx,int Nloc);


/*
INPUT:
 1. File location/ID for the magnification map
 2. A vector<double> of the profile half-light radii in units of RE (length of Nprof)
 3. The shape of the profile (could easily be a vector of time-dependent profiles)
 4. The number of sampled locations (Nloc)
 5. Two arrays of the x and y pixel indices to sample from (length of Nloc)
 6. A pointer to the CPU memory to store the output - size of Nprof*Nloc
 6. A pointer to the CPU memory to store the adjacent difference output - size of (Nprof-1)*Nloc


FUNCTION:
First, some memory needs to be allocated on the GPU:
1. To store the Fourier transform of the magnification map.
2. To store the Fourier transform of the convolution kernel (changes per loop iteration, see below).
3. To store the convolved map. This memory is also used to read-in the map and kernel before transforming them.
4. There is extra memory needed to run the CUFFT plans.
5. An 'LC' array of size Nprof x Nloc to store the light curves.

First, we get the Fourier transform of the map and store it on the GPU.
Inside a loop over the profile sizes we:
- create the profile according to the given shape
- create the corredponding kernel at the predefined memory location
- get the Fourier transform of the kernel
- multiply it with the Fourier transform of the map
- get inverse Fourier transform of the product
- sample the convolved map at the given pixels and store them in the LC array


OUTPUT:
The LC array with size Nloc*Nprof.
The array of differences between adjacent profiles for each light curve with size Nloc*(Nprof-1).

NOTES: 
- This code should ensure early on that there is enough memory on the GPU for it to run.
*/

void expanding_source(std::string map_id,std::vector<double> sizes,std::string shape,int Nloc,int* sample_loc_x,int* sample_loc_y,double* LC,double* DLC){

  // ############################################## Initialization ############################################################
  int Nprof = sizes.size();

  // Important definitions for the grids of blocks and blocks of threads
  dim3 block_mult(1000); // Nx/f
  dim3 grid_mult(10,5001); // f,Ny/2+1
  dim3 block_samp(1024);
  dim3 grid_samp((int) ceil(Nloc/1024));
    
  // We read the magnification map stored in the gerlumph format (map.bin and map_meta.dat)
  double dum_rein = 1.0;
  gerlumph::MagnificationMap map(map_id,dum_rein);
  int Nx = map.Nx;
  int Ny = map.Ny;
  double norm = Nx*Ny;
  //map.writeImageFITS("map.fits",10);
  //gerlumph::MagnificationMap dum_map = map; // A test map on the CPU
  
  // Calculate the maximum offset in pixels
  double max_size = sizes.back(); // in units of Rein
  int maxOffset = (int) ceil( (Nx/map.width)*max_size );
  gerlumph::Kernel kernel(map.Nx,map.Ny);

  // Create profile parameters
  std::map<std::string,std::string> profile_pars;
  profile_pars.insert(std::pair<std::string,std::string>("shape", shape));
  profile_pars.insert(std::pair<std::string,std::string>("rhalf", "dum"));
  profile_pars.insert(std::pair<std::string,std::string>("pixSizePhys", std::to_string(map.pixSizePhys)));
  profile_pars.insert(std::pair<std::string,std::string>("incl", "0"));
  profile_pars.insert(std::pair<std::string,std::string>("orient", "0"));
  // ##########################################################################################################################

  
  
  // ############################################## Memory Allocation on the GPU ##############################################
  printf("Used GPU memory:  %d (Mb)\n",(int) get_used_gpu_mem());

  
  // Allocate memory: for the sampled locations
  int* loc_x;
  hipMalloc((void**) &loc_x,Nloc*sizeof(int));
  hipMemcpy(loc_x,sample_loc_x,Nloc*sizeof(int),hipMemcpyHostToDevice);
  int* loc_y;
  hipMalloc((void**) &loc_y,Nloc*sizeof(int));
  hipMemcpy(loc_y,sample_loc_y,Nloc*sizeof(int),hipMemcpyHostToDevice);
  printf("Allocated memory: positions x and y, 2xNloc <int>: %d (Mb)\n",2*Nloc*4);

  // Allocate memory: for the Fourier transform of the map on the GPU (COMPLEX)
  hipfftDoubleComplex* Fmap_GPU;
  hipMalloc( (void**) &Fmap_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  if (cuda_error("Cuda error: Failed to allocate Fmap_GPU", false, __FILE__, __LINE__)) throw std::bad_alloc();
  printf("Allocated memory: Fourier transform of map, Nx(Ny/2+1) <double complex>: %d (Mb)\n",Nx*(Ny/2+1)*16);

  // Allocate memory: for the Fourier transform of the kernel on the GPU (COMPLEX)
  hipfftDoubleComplex* Fkernel_GPU;
  hipMalloc( (void**) &Fkernel_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  if (cuda_error("Cuda error: Failed to allocate Fkernel_GPU", false, __FILE__, __LINE__)) throw std::bad_alloc();
  printf("Allocated memory: Fourier transform of kernel, Nx(Ny/2+1) <double complex>: %d (Mb)\n",Nx*(Ny/2+1)*16);

  // Allocate memory: to be used for both the map and the convolved map on the GPU (REAL)
  hipfftDoubleReal* any_map_GPU;
  hipMalloc((void**) &any_map_GPU, Nx*Ny*sizeof(hipfftDoubleReal));
  if (cuda_error("Cuda error: Failed to allocate any_map_GPU", false, __FILE__, __LINE__)) throw std::bad_alloc();
  printf("Allocated memory: Any real map, Nx*Ny <double>: %d (Mb)\n",Nx*Ny*8);

  // Allocate memory: for the final LC array
  double* d_LC;
  hipMalloc(&d_LC,Nprof*Nloc*sizeof(double));
  double* d_DLC;
  hipMalloc(&d_DLC,(Nprof-1)*Nloc*sizeof(double));
  printf("Allocated memory: Light curves, (2Nprof-1)*Nloc <double>: %d (Mb)\n",(2*Nprof-1)*Nloc*8);

  // Create CUFFT plans
  hipfftResult result;
  size_t plan_size;

  hipfftHandle plan_r2c;
  result = hipfftPlan2d(&plan_r2c,Nx,Ny,HIPFFT_D2Z);
  if (cuda_error("Cuda error: Failed to create plan", false, __FILE__, __LINE__)) throw std::bad_alloc();
  result = hipfftGetSize2d(plan_r2c,Nx,Ny,HIPFFT_D2Z,&plan_size);
  printf("Allocated memory: plan R2C: %d (Mb)\n",(int) plan_size);

  hipfftHandle plan_c2r;
  result = hipfftPlan2d(&plan_c2r,Nx,Ny,HIPFFT_Z2D);
  if (cuda_error("Cuda error: Failed to create plan", false, __FILE__, __LINE__)) throw std::bad_alloc();  
  result = hipfftGetSize2d(plan_c2r,Nx,Ny,HIPFFT_Z2D,&plan_size);
  printf("Allocated memory: plan C2R: %d (Mb)\n",(int) plan_size);


  printf("Used GPU memory:  %d (Mb)\n",(int) get_used_gpu_mem());
  // ##########################################################################################################################



  // ############################################## Operations of the GPU #####################################################
  // Do the Fourier transform of the emap and store it on the GPU
  hipMemcpy( any_map_GPU, map.data, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
  myfft2d_r2c(&plan_r2c,any_map_GPU,Fmap_GPU);

  // Loop over the kernels
  for(int k=0;k<Nprof;k++){
    std::cout << "Profile: " << k << std::endl;

    // Create profile and kernel
    profile_pars["rhalf"] = std::to_string(sizes[k]*map.pixSizePhys*map.Nx/map.width);
    gerlumph::BaseProfile* profile = gerlumph::FactoryProfile::getInstance()->createProfileFromHalfRadius(profile_pars);
    kernel.setKernel(profile);
    delete(profile);

    // Fourier transform of kernel
    hipMemcpy( any_map_GPU, kernel.data, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
    myfft2d_r2c(&plan_r2c,any_map_GPU,Fkernel_GPU);

    // Multiply Fourier transforms of map and kernel
    kernelMultiplyFFTs<<<grid_mult,block_mult>>>(Fmap_GPU,Fkernel_GPU,norm);
    hipDeviceSynchronize();

    // Get inverse Fourier transform of product
    myfft2d_c2r(&plan_c2r,Fkernel_GPU,any_map_GPU);

    // Transfer convolved map to CPU and write image.
    //hipMemcpy(dum_map.data,convmap_GPU,Nx*Ny*sizeof(double),hipMemcpyDeviceToHost);
    //dum_map.writeImageFITS("conv_"+std::to_string(k)+".fits",10);

    // Sample convolved map
    sampleConvmap<<<grid_samp,block_samp>>>(any_map_GPU,d_LC,d_DLC,k,Nprof,loc_x,loc_y,Nx,Nloc);
  }
  // ##########################################################################################################################



  // ############################################## Fetch light curves ########################################################
  hipMemcpy(LC,d_LC,Nprof*Nloc*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(DLC,d_DLC,(Nprof-1)*Nloc*sizeof(double),hipMemcpyDeviceToHost);
  // ##########################################################################################################################


  
  // ############################################## Cleanup ###################################################################
  hipFree(Fkernel_GPU);
  hipFree(Fmap_GPU);
  hipFree(any_map_GPU);
  hipFree(loc_x);
  hipFree(loc_y);
  hipFree(d_LC);
  hipfftDestroy(plan_r2c);
  hipfftDestroy(plan_c2r);
  // ##########################################################################################################################
}


int myfft2d_r2c(hipfftHandle* plan,hipfftDoubleReal* data_GPU,hipfftDoubleComplex* Fdata_GPU){
  // Do the fourier transform on the GPU
  hipfftResult result = hipfftExecD2Z(*plan, data_GPU, Fdata_GPU);
  if (cuda_error("Cuda error: Failed to execut plan", false, __FILE__, __LINE__)) std::runtime_error("CUFFT Error: unable to execute plan");
  hipDeviceSynchronize();
  return 0;
}


int myfft2d_c2r(hipfftHandle* plan, hipfftDoubleComplex* Fdata_GPU, hipfftDoubleReal* data_GPU){
  // Do the inverse fourier transform on the GPU
  hipfftResult result = hipfftExecZ2D(*plan, Fdata_GPU, data_GPU);
  if (cuda_error("Cuda error: Failed to execut plan", false, __FILE__, __LINE__)) std::runtime_error("CUFFT Error: unable to execute plan");
  hipDeviceSynchronize();
  return 0;
}


__global__ void kernelMultiplyFFTs(hipfftDoubleComplex* Fmap,hipfftDoubleComplex* Fkernel,double norm){
  unsigned long int i = (blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x + threadIdx.x; // thread ID
  hipfftDoubleReal dum1 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].x - Fmap[i].y*Fkernel[i].y);
  hipfftDoubleReal dum2 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].y + Fmap[i].y*Fkernel[i].x);
  Fkernel[i].x = dum1/norm;
  Fkernel[i].y = dum2/norm;
}


__global__ void sampleConvmap(hipfftDoubleReal* convmap,double* LC,double* DLC,int k,int Nprof,int* loc_x,int* loc_y,int Nx,int Nloc){
  // Stores the light curve in sizes of Nprof
  //unsigned int id = threadIdx.x;
  unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
  if( id<Nloc ){
    unsigned int i = loc_y[id];
    unsigned int j = loc_x[id];
    unsigned int index = i*Nx+j;
    LC[id*Nprof+k] = convmap[index];

    if( k>0 ){
      DLC[id*Nprof+k-1] = LC[id*Nprof+k] - LC[id*Nprof+k-1];
    }
  }
}








