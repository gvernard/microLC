#include "hip/hip_runtime.h"
#include "chi2_functions.hpp"


__global__ void kernelChi2(float* d_mua, float* d_mub, float* d_like_dum, int Nwave, int Nlocs);
__constant__ float const_f_obs[64];
__constant__ float const_df_obs[64];



/*******************************************************************************
calculate the chi-squared likelihood on the GPU

\param like_all -- pointer to array of likelihoods
\param f_obs_all -- pointer to array of observation flux ratios
                    assumes array is of the form [filter1, ... filterNwave]
                    where each filter is of the form [loc1, ... locNlocs]
\param df_obs -- pointer to array of errors on the observations
                 assumes a single value for the error per filter
                 i.e. it is of the form [filter1, ... filterNwave]
\param h_mua -- pointer to array of simulated fluxes for image 1
                see f_obs_all for its structure
\param h_mub -- pointer to array of simulated fluxes for image 2
                see f_obs_all for its structure
\param Nlocs -- number of locations simulated for each image.
                assumed to be the same for image 1 and image 2
\param Nwave -- number of wavelengths (filters)
\param Nratios -- number of observations (timestamps)
*******************************************************************************/
void getChi2Cuda(double* like_all, float** f_obs_all, float* df_obs, float** h_mua, float** h_mub, int Nlocs, int Nwave, int Nratios)
{
  hipError_t err;

  // Size of the shared memory required in the kernel is always Nwave * Nthreads * sizeof(float)
  // i.e. this is the chunk of muA locations for all wavelengths.
  // I need to generally keep this below 48KB.
  int Ngrid;
  int Nthreads;
  //ensure that grid and block size are such that memory is below 30KB
  setGridThreads(Ngrid, Nthreads, Nwave); 
  printf("Shared memory occupancy: %d / 48000\n", Nwave * Nthreads * sizeof(float));
  printf("Ngrid / Nthreads = %d / %d\n", Ngrid, Nthreads);


  // Allocate memory on device
  // inputs:
  float* d_mua;
  float* d_mub;
  hipMalloc(&d_mua, Nwave * Nlocs * sizeof(float));
  hipMalloc(&d_mub, Nwave * Nlocs * sizeof(float));

  // output:
  int Nlike = Ngrid * Nthreads;
  float* h_like_dum = (float*) malloc(Nlike * sizeof(float));
  float* d_like_dum;
  hipMalloc(&d_like_dum, Nlike * sizeof(float));
  err = hipGetLastError();
  if(err != hipSuccess)
  {
    fprintf(stderr, "Error: %s - in \"memory allocation\" \n", hipGetErrorString(err));
  }

  // Transfer (input) memory from host to device
  for(int k = 0; k < Nwave; k++)
  {
    hipMemcpy(d_mua + k * Nlocs, h_mua[k], Nlocs * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mub + k * Nlocs, h_mub[k], Nlocs * sizeof(float), hipMemcpyHostToDevice);
  }
  //assumes we have 64 or less filters (see definition of const_df_obs at start of file)
  hipMemcpyToSymbol(HIP_SYMBOL(const_df_obs), (void*)df_obs, Nwave * sizeof(float), 0);
  err = hipGetLastError();
  if(err != hipSuccess)
  {
    fprintf(stderr, "Error: %s - in \"memory transfer to device\" \n", hipGetErrorString(err));
  }


  // Loop over f_obs (Nratios times)
  dim3 grid(Ngrid);
  dim3 threads(Nthreads);
  int Nmem = Nwave * Nthreads;
  //for every timestamp
  for(int i = 0; i < Nratios; i++)
  {
    // Transfer f_obs to constant memory
    //assumes we have 64 or less filters (see definition of const_f_obs at start of file)
    hipMemcpyToSymbol(HIP_SYMBOL(const_f_obs), (void*)f_obs_all[i], Nwave * sizeof(float), 0);

    // Execute GPU kernels
    kernelChi2<<<grid, threads, Nmem * sizeof(float)>>>(d_mua, d_mub, d_like_dum, Nwave, Nlocs);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if(err != hipSuccess)
    {
      fprintf(stderr,"Error: %s - in \"kernelChi2\" \n", hipGetErrorString(err));
    }

    // Get memory from device
    hipMemcpy(h_like_dum, d_like_dum, Nlike * sizeof(float), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
      fprintf(stderr,"Error: %s - in \"memory transfer from device\" \n", hipGetErrorString(err));
    }

    // Add all the likelihoods in h_like_dum
    double like = 0.0;
    for(int j = 0; j < Nlike; j++)
    {
      like += h_like_dum[j];
    }
    like_all[i] = like;

    std::cout << like_all[i] << std::endl;
  }

  free(h_like_dum);
}



/*******************************************************************************
calculate the chi-squared likelihood on the GPU for a single timestamp

\param d_mua -- pointer to array of simulated fluxes for image 1
\param d_mub -- pointer to array of simulated fluxes for image 2
\param d_like_dum -- pointer to array of likelihoods
\param Nwave -- number of wavelengths (filters)
\param Nlocs -- number of locations simulated for each image.
                assumed to be the same for image 1 and image 2
*******************************************************************************/
__global__ void kernelChi2(float* d_mua, float* d_mub, float* d_like_dum, int Nwave, int Nlocs)
{
  unsigned int t         = threadIdx.x;
  unsigned int Nthreads  = blockDim.x;
  unsigned int thread_id = blockIdx.x * Nthreads + t;
  unsigned int Nblocks   = gridDim.x;

  // Allocate shared memory
  extern __shared__ float mua[];

  //this uses float, but GPUs are capable of double precision now
  //not sure if the greater precision from doubles really matters to us though
  float like = 0.0;
  // Loop over all of muA, reading it block by block
  for(int j = 0; j < Nblocks; j++)
  {
    // Each thread reads Nwave entries from muA into shared memory (different wavelengths from the same magmap location)
    for(int k = 0; k < Nwave; k++)
    {
      mua[k * Nthreads + t] = d_mua[k * Nlocs + Nthreads * j + t];
    }
    __syncthreads();

    // Each thread combines its unique muB value (thread_id) with the muA values currently into shared memory
    for(int i = 0; i < Nthreads; i++)
    {
      float fac = 0.0;
      for(int k = 0; k < Nwave; k++)
      {
        float fsim = mua[k * Nthreads + i]/d_mub[k * Nlocs + thread_id];
        float dum = (const_f_obs[k] - fsim) / const_df_obs[k];
        fac += dum * dum;
      }
      like += exp(-fac / 2);
    }
  }
  d_like_dum[thread_id] = like;
}



/*******************************************************************************
set the number of blocks in the grid, and the number of threads in a block,
for a given number of filters
ensures that grid and block size are such that memory in a block is below 30KB

\param Ngrid -- reference to number of blocks per grid
\param Nthreads -- reference to number of threads per block
\param Nwave -- number of filters (wavelengths)
*******************************************************************************/
void setGridThreads(int& Ngrid, int& Nthreads, int Nwave)
{
  Ngrid = 10;
  Nthreads = 1000;
  int shared_mem_size = Nthreads * Nwave * sizeof(float);

  while (shared_mem_size > 30000)
  {
    Ngrid *= 2;
    Nthreads /= 2;
    shared_mem_size = Nthreads * Nwave * sizeof(float);
  }
}



/*******************************************************************************
calculate the chi-squared likelihood on the CPU

\param like_all -- pointer to array of likelihoods
\param f_obs_all -- pointer to array of observation flux ratios
\param df_obs -- pointer to array of errors on the observations
\param h_mua -- pointer to array of simulated fluxes for image 1
\param h_mub -- pointer to array of simulated fluxes for image 2
\param Nlocs -- number of locations simulated for each image.
                assumed to be the same for image 1 and image 2
\param Nwave -- number of wavelengths (filters)
\param Nratios -- number of observations (timestamps)
*******************************************************************************/
void getChi2CudaCPU(double* like_all, float** f_obs_all, float* df_obs, float** h_mua, float** h_mub, int Nlocs, int Nwave, int Nratios)
{

  //for every timestamp
  for(int q = 0; q < Nratios; q++)
  {
    double like = 0.0;
    //for every simulated lightcurve
    for(int i = 0; i < Nlocs; i++)
    {
      for(int j = 0; j < Nlocs; j++)
      {
        double chi2 = 0.0;
        //for every filter
        for(int k = 0; k < Nwave; k++)
        {
          double fsim = h_mua[k][i] / h_mub[k][j];
          double dum = (f_obs_all[q][k] - fsim) / df_obs[k];
          chi2 += dum * dum;
        }
        //add to the likelihood for this timestamp
        //should we calculate the log likelihood instead to avoid
        //floating point precision loss?
        //in addition, it's not immediately obvious that adding the likelihood from
        //every filter together is the right thing to do per timestamp
        //although after some thought, so long as the simulations account for the different
        //sizes per time stamp, maybe it is?
        like += exp(-chi2 / 2);
      }
    }
    //set the likelihood for this timestamp
    like_all[q] = like;
    std::cout << like_all[q] << std::endl;
  }
}

